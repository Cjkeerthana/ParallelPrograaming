#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<time.h>
#include<sys/time.h>
#include<hip/hip_runtime_api.h>
#include<hipblas.h>
#include"mat_mul.h"

double *d_A, *d_B, *d_C;
size_t size_A, size_B, size_C;
hipblasHandle_t handle;
double t_start_cpy, t_end_cpy;
hipEvent_t t_start_calc, t_end_calc;
const double alf = 1.0;
const double bet = 0.0;
const double *alpha = &alf;
const double *beta = &bet;

void cuda_initialize(double* A, size_t m, size_t n, size_t p, int rank, double* t_cpy)
{

 int deviceCount;
 hipGetDeviceCount(&deviceCount);
 int device_id = rank % deviceCount;
 hipSetDevice(device_id);
 
 size_A = m*p*sizeof(double);
 size_B = p*n*sizeof(double);
 size_C = m*n*sizeof(double);

 hipMalloc((void **)&d_A, size_A);
 hipMalloc((void **)&d_B, size_B);
 hipMalloc((void **)&d_C, size_C);
 
 hipblasCreate(&handle);

 t_start_cpy = seconds();
 hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
 t_end_cpy = seconds();
 
 *t_cpy += t_end_cpy - t_start_cpy;

}

void cuda_vector_mul(double* B, double** C, size_t m, size_t n, size_t p, double* t_cpy, double* t_calc){

 t_start_cpy = seconds();
 hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
 t_end_cpy = seconds();

 *t_cpy += t_end_cpy - t_start_cpy;

 //hipDeviceSynchronize();
 //printf("Calculating in GPU\n");
 //hipEventRecord(t_start_calc, 0);
 hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, p, alpha, d_B, n, d_A, p, beta, d_C, n);
 //hipDeviceSynchronize();
 //hipEventRecord(t_end_calc, 0);
 
 //float calc_time = 0.0f;
 //hipEventElapsedTime(&calc_time, t_start_calc, t_end_calc);
 //*t_calc += (double) calc_time;

 t_start_cpy = seconds();
 hipMemcpy(*C, d_C, size_C, hipMemcpyDeviceToHost); 
 t_end_cpy = seconds();

 *t_cpy += t_end_cpy - t_start_cpy;

}

void cuda_stop(){
 
 hipblasDestroy(handle);
 
 hipFree(d_A);
 hipFree(d_B);
 hipFree(d_C);
}
